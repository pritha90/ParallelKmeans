#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kmeans.h"


__global__ static
void find_nearest_cluster(int dimension,
                          int numObjs,
                          int numClusters,
                          float *data,
                          float *DEVICEcenter,
                          int *newmembership)
{
    
    int dataId = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (dataId < numObjs) {
        int   i,j;
        float distance, min_dist = 0.0;
        
        for(i=0; i<dimension; i++)
        {
            min_dist +=(data[numObjs * i + dataId] - DEVICEcenter[numClusters * i]) *
            (data[numObjs * i + dataId] - DEVICEcenter[numClusters * i]);
        newmembership[dataId]=0;
        }
            
        for (j=1; j<numClusters; j++) {
            distance = 0.0;
            for(i=0; i<dimension; i++)
            {
                distance +=(data[numObjs * i + dataId] - DEVICEcenter[numClusters * i + j]) *
                (data[numObjs * i + dataId] - DEVICEcenter[numClusters * i + j]);
            }
            if (distance < min_dist) {
                min_dist = distance;
                newmembership[dataId] = j;
            }
        }
    
         __syncthreads();
    }
}

float** cuda_kmeans(float **data, int dimension, int numObjs, int numClusters, float threshold                 , int *membership, int *num_iterations)
{
    int      i, j, numiterations=0;
    int     *clustersize;
    float    delta, **center, **clustersum;
    float  **datatranspose;
    float  **centertranspose;
    float *DEVICEdata;
    float *DEVICEcenter;
    int *DEVICEmembership;
    int *newmembership;

    //center    = (float**) malloc(numClusters *sizeof(float*));
    malloc2D(center,numClusters,dimension);
    malloc2D(datatranspose, dimension, numObjs);
    for (i = 0; i < dimension; i++) {
        for (j = 0; j < numObjs; j++) {
            datatranspose[i][j] = data[j][i];
        }
    }
    
    malloc2D(centertranspose, dimension, numClusters);
    for (i = 0; i < dimension; i++) {
        for (j = 0; j < numClusters; j++) {
            centertranspose[i][j] = datatranspose[i][j];
        }
    }
    
    clustersize = (int*) calloc(numClusters, sizeof(int));
    malloc2D(clustersum, numClusters, dimension);
    for (i=0; i<numClusters; i++){
        for(j=0; j<dimension; j++){
             clustersum[i][j] = 0.0;
    }
    }        

    newmembership = (int*) calloc(numObjs, sizeof(int));
    
    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =(numObjs + numThreadsPerClusterBlock - 1) /numThreadsPerClusterBlock;

    hipMalloc(&DEVICEdata, numObjs*dimension*sizeof(float));
    hipMalloc(&DEVICEcenter, numClusters*dimension*sizeof(float));
    hipMalloc(&DEVICEmembership, numObjs*sizeof(int));
    hipMemcpy(DEVICEdata, datatranspose[0],
                         numObjs*dimension*sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(DEVICEmembership, membership,
    //                     numObjs*sizeof(int), hipMemcpyHostToDevice);
    
    do {
        hipMemcpy(DEVICEcenter, centertranspose[0],numClusters*dimension*sizeof(float), hipMemcpyHostToDevice);
        
        find_nearest_cluster
        <<< numClusterBlocks, numThreadsPerClusterBlock>>>
        (dimension, numObjs, numClusters,
         DEVICEdata, DEVICEcenter, DEVICEmembership);
        
        hipDeviceSynchronize();
        
        hipMemcpy(newmembership, DEVICEmembership, numObjs*sizeof(int), hipMemcpyDeviceToHost);
        
        delta = 0.0;
        for(i=0; i<numObjs; i++) // dont recalculate new centers afresh
        {
            if(numiterations == 0){
                delta =float(numObjs);
                membership[i]=newmembership[i];
                clustersize[membership[i]]++;
                 for(j=0; j<dimension; j++){
                clustersum[membership[i]][j] += data[i][j];
              }
            }
            
            else if(membership[i] != newmembership[i]){
                delta += 1.0;
                clustersize[newmembership[i]] ++;
                clustersize[membership[i]] --;
                for(j=0; j<dimension; j++){
                    clustersum[newmembership[i]][j] += data[i][j];
                    clustersum[membership[i]][j] -= data[i][j];
                }
                membership[i] = newmembership[i];
            }
        }
        
    
        for (i=0; i<numClusters; i++) {
            for (j=0; j<dimension; j++) {
                if (clustersize[i] > 0)
                    centertranspose[j][i] = clustersum[i][j] / clustersize[i];
            }
        }        
        delta /= numObjs;
    } while (delta > threshold && numiterations++ < 30);
    
    *num_iterations = numiterations + 1;
    
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < dimension; j++) {
            center[i][j] = centertranspose[j][i];
        }
    }
    
    hipFree(DEVICEdata);
    hipFree(DEVICEcenter);
    hipFree(DEVICEmembership);
    
    free(datatranspose[0]);
    free(datatranspose);
    free(centertranspose[0]);
    free(centertranspose);
    free(clustersum[0]);
    free(clustersum);
    free(clustersize);
    free(newmembership);
    return center;
}
